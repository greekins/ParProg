#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"

__global__ void VectorAddKernel(float *a, float *b, float *c, int numElements) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < numElements) {
		c[i] = a[i] + b [i];
	}
}

void cudaVectorAdd(float *a, float *b, float *c, int numElements) {
	clock_t start = clock();

	// TODO: Implement a parallel vector addition on CUDA
	size_t size = numElements * sizeof(float);
	float *d_a, *d_b, *d_c;
	
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	int blockDim = 512;
	int gridDim = (numElements + blockDim -1) / blockDim;
	VectorAddKernel<<<gridDim, blockDim>>>(d_a, d_b, d_c, numElements);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	float diff = float(clock() - start) / CLOCKS_PER_SEC;
	printf("CUDA: %.3lf seconds\n", diff);
}

void fillRandomArray(float *a, int numElements) {
	for (int i = 0; i < numElements; i++) {
		a[i] = rand() / (float)RAND_MAX;
	}
}

void verifyResults(float *a, float *b, float *c, int numElements) {
	for (int i = 0; i < numElements; i++) {
        if (fabs(a[i] + b[i] - c[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
}

void sequentialVectorAdd(float *a, float *b, float *c, int numElements) {
	clock_t start = clock();

	for (int i = 0; i < numElements; i++) {
		c[i] = a[i] + b[i];
	}

	float diff = float(clock() - start) / CLOCKS_PER_SEC;
	printf("Sequential: %.3lf seconds\n", diff);
}

int main() {
	int N = 10000000;
	size_t size = N * sizeof(float);

	float *h_a = (float *)malloc(size);
	handleAllocationError(h_a);
	fillRandomArray(h_a, N);
	
	float *h_b = (float *)malloc(size);
	handleAllocationError(h_b);
	fillRandomArray(h_b, N);
	
	float *h_c = (float *)malloc(size);
	handleAllocationError(h_c);

	cudaVectorAdd(h_a, h_b, h_c, N);
	verifyResults(h_a, h_b, h_c, N);

	sequentialVectorAdd(h_a, h_b, h_c, N);

	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
